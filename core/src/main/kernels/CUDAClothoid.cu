
#include <hip/hip_runtime.h>
#include "math.h"

__device__ void getCircleTrajectory(int nbPoint, float curvature, float orientation, float xRobot, float yRobot, float* xOutput, float* yOutput, float* orientationOutput, float* curvatureOutput)
{
  float radius = 1000.f / curvature;
  float xDelta = cosf(orientation + 3.1415f/2.f) * radius;
  float yDelta = sinf(orientation + 3.1415f/2.f) * radius;
  float xCenter = xRobot + xDelta;
  float yCenter = yRobot + yDelta;
  float angle = 0.02f * curvature;
  float cosv = cosf(angle);
  float sinv = sinf(angle);
  float tmp = xDelta;
  xDelta = cosv * xDelta - sinv * yDelta;
  yDelta = sinv * tmp + cosv * yDelta;
  *xOutput = xCenter - xDelta;
  *yOutput = yCenter - yDelta;
  *orientationOutput = orientation + (nbPoint+1) * angle;
  *curvatureOutput = curvature;
}

__device__ void getStraightTrajectory(int nbPoint, float orientation, float xRobot, float yRobot, float* xOutput, float* yOutput, float* orientationOutput, float* curvatureOutput)
{
  float cosv = cosf(orientation);
  float sinv = sinf(orientation);
  float d = (float)(nbPoint + 1) * 20.f;
  *xOutput = xRobot + d * cosv;
  *yOutput = yRobot + d * sinv;
  *curvatureOutput = 0;
  *orientationOutput = orientation;
}

__device__ void compute(int nbPoint, float* xUnitary, float* yUnitary, int tentacleSpeed, float tentacleSquaredRootSpeed, float tentacleInitialCurvature, int16_t tentaclePositive, int16_t tentacleBack, int16_t tentacleStop, float xRobot, float yRobot, float orientationRobot, float curvatureRobot, int16_t goingForwardRobot, float* xOutput, float* yOutput, float* orientationOutput, float* curvatureOutput, int16_t* goingForwardOutput)
{
  if(tentacleBack)
    orientationRobot += 3.1415f;

  if(tentacleBack || tentacleStop)
    curvatureRobot = tentacleInitialCurvature;

  *goingForwardOutput = tentacleBack != goingForwardRobot;
  if(tentacleSpeed == 0)
  {
    if(curvatureRobot < 0.0001f && curvatureRobot > -0.0001f)
      getStraightTrajectory(nbPoint, orientationRobot, xRobot, yRobot, xOutput, yOutput, orientationOutput, curvatureOutput);
    else
      getCircleTrajectory(nbPoint, curvatureRobot, orientationRobot, xRobot, yRobot, xOutput, yOutput, orientationOutput, curvatureOutput);
  }
  else
  {
    float coeff = 1.f / tentacleSquaredRootSpeed;
    float sDepart = curvatureRobot * coeff;
    if(tentacleSpeed < 0)
      sDepart = -sDepart;

    int pointDepart = (int) ((sDepart / 0.02f) + 500.f - 1.f + 0.5f);
    float orientationClotho = sDepart * sDepart;
    if(tentacleSpeed < 0)
      orientationClotho = -orientationClotho;
    float base = orientationRobot  - orientationClotho;
    float cosv = cosf(base);
    float sinv = sinf(base);
    sDepart += ((float)(nbPoint+1) * tentacleSquaredRootSpeed * 0.02f);

    *xOutput = (float)(xUnitary[(int)(pointDepart + (int)tentacleSquaredRootSpeed * (nbPoint + 1))] - xUnitary[pointDepart]) * coeff;
    *yOutput = (float)(yUnitary[(int)(pointDepart + (int)tentacleSquaredRootSpeed * (nbPoint + 1))] - yUnitary[pointDepart]) * coeff;
    if(tentacleSpeed < 0)
      *yOutput = -*yOutput;
    float tmp = *xOutput;
    *xOutput = cosv*(*xOutput) - sinv*(*yOutput) + xRobot;
    *yOutput = sinv*tmp + cosv*(*yOutput) + yRobot;
    *curvatureOutput = sDepart * tentacleSquaredRootSpeed;
    *orientationOutput = sDepart * sDepart;
    if(tentacleSpeed < 0)
    {
      *orientationOutput = -*orientationOutput;
      *curvatureOutput = -*curvatureOutput;
    }
    *orientationOutput += base;
  }
}

extern "C"
__global__ void kernelFunc(float* xUnitary, float* yUnitary, int* tentacleSpeed, float* tentacleSquaredRootSpeed, float* tentacleInitialCurvature, int16_t* tentaclePositive, int16_t* tentacleBack, int16_t* tentacleStop, float* xRobot, float* yRobot, float* orientationRobot, float* curvatureRobot, int16_t* goingForwardRobot, float* xOutput, float* yOutput, float* orientationOutput, float* curvatureOutput, int16_t* goingForwardOutput)
{
    int nbPoint = blockIdx.y;
    int tentaculeNumber = blockIdx.x;

    compute(nbPoint, xUnitary, yUnitary, tentacleSpeed[tentaculeNumber], tentacleSquaredRootSpeed[tentaculeNumber], tentacleInitialCurvature[tentaculeNumber], tentaclePositive[tentaculeNumber], tentacleBack[tentaculeNumber], tentacleStop[tentaculeNumber], xRobot[0], yRobot[0], orientationRobot[0], curvatureRobot[0], goingForwardRobot[0], &xOutput[5*tentaculeNumber + nbPoint], &yOutput[5*tentaculeNumber + nbPoint], &orientationOutput[5*tentaculeNumber + nbPoint], &curvatureOutput[5*tentaculeNumber + nbPoint], &goingForwardOutput[5*tentaculeNumber + nbPoint]);
}
