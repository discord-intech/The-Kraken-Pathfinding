#include "hip/hip_runtime.h"
extern "C"
void getCircleTrajectory(int nbPoint, float curvature, float orientation, float xRobot, float yRobot, float* xOutput, float* yOutput, float* orientationOutput, float* curvatureOutput)
{
  float radius = 1000.f / curvature;
  float xDelta = cos(orientation + 3.1415f/2.f) * radius;
  float yDelta = sin(orientation + 3.1415f/2) * radius;
  float xCenter = xRobot + xDelta;
  float yCenter = yRobot + yDelta;
  float angle = (nbPoint+1) * 0.02f * curvature;
  float cosv = cos(angle);
  float sinv = sin(angle);
  float tmp = xDelta;
  xDelta = cosv * xDelta - sinv * yDelta;
  yDelta = sinv * tmp + cosv * yDelta;
  *xOutput = xCenter - xDelta;
  *yOutput = yCenter - yDelta;
  *orientationOutput = orientation + angle;
  *curvatureOutput = curvature;
}

extern "C"
void getStraightTrajectory(int nbPoint, float orientation, float xRobot, float yRobot, float* xOutput, float* yOutput, float* orientationOutput, float* curvatureOutput)
{
  float cosv = cos(orientation);
  float sinv = sin(orientation);
  float d = (nbPoint + 1) * 0.02f;
  *xOutput = xRobot + d * cosv;
  *yOutput = yRobot + d * sinv;
  *curvatureOutput = 0;
  *orientationOutput = orientation;
}


extern "C"
void compute(int nbPoint, float[] xUnitary, float[] yUnitary, int tentacleSpeed, float tentacleSquaredRootSpeed, float tentacleInitialCurvature, short tentaclePositive, short tentacleBack, short tentacleStop, float xRobot, float yRobot, float orientationRobot, float curvatureRobot, short goingForwardRobot, float* xOutput, float* yOutput, float* orientationOutput, float* curvatureOutput, short* goingForwardOutput)
{
  if(tentacleBack)
    orientationRobot += 3.1415f;

  if(tentacleBack || tentacleStop)
    curvatureRobot = tentacleInitialCurvature;

  *goingForwardOutput = tentacleBack != goingForwardRobot;
  if(tentacleSpeed == 0)
  {
    if(curvatureRobot == 0)
      getStraightTrajectory(nbPoint, orientationRobot, yRobot, yRobot, xOutput, yOutput, orientationOutput, curvatureOutput);
    else
      getCircleTrajectory(nbPoint, curvatureRobot, orientationRobot, xRobot, yRobot, xOutput, yOutput, orientationOutput, curvatureOutput);
  }
  else
  {
    float coeff = 1.f / tentacleSquaredRootSpeed;
    float sDepart = curvatureRobot * coeff;
    if(tentacleSpeed < 0)
      sDepart = -sDepart;

    int pointDepart = (int) ((sDepart / 0.02f) + 500.f - 1.f + 0.5f);
    float orientationClotho = sDepart * sDepart;
    if(tentacleSpeed < 0)
      orientationClotho = -orientationClotho;
    float base = orientationRobot  - orientationClotho;
    float cosv = cos(base);
    float sinv = sin(base);
    sDepart += (nbPoint+1) * tentacleSquaredRootSpeed * 0.02f;

    xOutput = (xUnitary[pointDepart + tentacleSquaredRootSpeed * (nbPoint + 1)] - xUnitary[pointDepart]) * coeff;
    yOutput = (yUnitary[pointDepart + tentacleSquaredRootSpeed * (nbPoint + 1)] - yUnitary[pointDepart]) * coeff;
    if(tentacleSpeed < 0)
      *yOutput = -*yOutput;
    float tmp = *xOutput;
    *xOutput = cosv*(*xOutput) - sinv*(*yOutput) + xRobot;
    *yOutput = sinv*tmp + cosv*(*yOutput) + yRobot;
    *curvatureOutput = sDepart * tentacleSquaredRootSpeed;
    *orientationOutput = sDepart * sDepart;
    if(tentacleSpeed < 0)
      *orientationOutput = -*orientationOutput;
    *orientationOutput += base;
  }
}

extern "C"
__global__ void kernelFunc(float* xUnitary, float* yUnitary, int* tentacleSpeed, float* tentacleSquaredRootSpeed, float* tentacleInitialCurvature, short* tentaclePositive, short* tentacleBack, short* tentacleStop, float* xRobot, float* yRobot, float* orientationRobot, float* curvatureRobot, short* goingForwardRobot, float* xOutput, float* yOutput, float* orientationOutput, float* curvatureOutput, short* goingForwardOutput)
{
    nbPoint = blockIdx.y;

    compute(nbPoint, xUnitary, yUnitary, tentacleSpeed[0], tentacleSquaredRootSpeed[0], tentacleInitialCurvature[0], tentacleInitialCurvature[0], tentaclePositive[0], tentacleBack[0], tentacleStop[0], xRobot[0], yRobot[0], orientationRobot[0], curvatureRobot[0], goingForwardRobot[0], &xOutput[blockIdx.x * 5 + nbPoint], &yOutput[blockIdx.x * 5 + nbPoint], &orientationOutput[blockIdx.x * 5 + nbPoint], &curvatureOutput[blockIdx.x * 5 + nbPoint], &goingForwardOutput[blockIdx.x * 5 + nbPoint])

}